#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN  // This tells Catch to provide a main() - only do this in one cpp file
#include "catch.hpp"
#include "stojanovski.cuh"


template < typename T >
__global__ void call_reduce( T * data, uint32_t const n )
{
    stoja::reduce( data, n );
}

struct wrap_reduce
{
    template < typename T >
    void operator () ( T * data, uint32_t const n ) const
    {
        auto const size = n * sizeof( T );

        T * dev_data;
        hipMalloc( (void **)&dev_data, size );
        REQUIRE( hipGetLastError() == hipSuccess );
        hipMemcpy( dev_data, data, size, hipMemcpyHostToDevice );
        REQUIRE( hipGetLastError() == hipSuccess );

        call_reduce<<< 1, n >>>( dev_data, n );
        REQUIRE( hipGetLastError() == hipSuccess );

        hipMemcpy( data, dev_data, size, hipMemcpyDeviceToHost );
        REQUIRE( hipGetLastError() == hipSuccess );
        hipFree( dev_data );
        REQUIRE( hipGetLastError() == hipSuccess );
    }
};



SCENARIO( "The min and max of several values is computed", "[max][util]")
{
  using namespace stoja;

  GIVEN( "Two uint32_t vals" )
  {
  	uint32_t x, y;

    WHEN( "The points are identical" )
    {
      x = 7u;
      y = 7u;

      THEN( "The max function returns their shared value" )
      {
        REQUIRE( max( x, y ) == 7u );
      }

      THEN( "The min function returns their shared value" )
      {
        REQUIRE( min( x, y ) == 7u );
      }
    }

    WHEN( "The first value is larger" )
    {
      x = 8u;
      y = 7u;

      THEN( "The max function returns the first value" )
      {
        REQUIRE( max( x, y ) == x );
      }

      THEN( "The min function returns the second value" )
      {
        REQUIRE( min( x, y ) == y );
      }
    }

    WHEN( "The first value is smaller" )
    {
      x = 6u;
      y = 7u;

      THEN( "The max function returns the second value" )
      {
        REQUIRE( max( x, y ) == y );
      }

      THEN( "The min function returns the first value" )
      {
        REQUIRE( min( x, y ) == x );
      }
    }
  }

    GIVEN( "Four uint32_t vals" )
    {
        uint32_t w, x, y, z;

        WHEN( "The points are identical" )
        {
            w = 7u;
            x = 7u;
            y = 7u;
            z = 7u;

            THEN( "The max function returns their shared value" )
            {
                REQUIRE( max( w, x, y, z ) == 7u );
            }
        }

        WHEN( "The points are all distinct" )
        {
            w = 7u;
            x = 8u;
            y = 9u;
            z = 6u;

            THEN( "The max function returns the largest" )
            {
                REQUIRE( max( w, x, y, z ) == y );
            }
        }

        WHEN( "The values are grouped in pairs" )
        {
            w = 7u;
            x = 7u;
            y = 9u;
            z = 9u;

            THEN( "The max function returns the largest" )
            {
                REQUIRE( max( w, x, y, z ) == z );
            }
        }
    }
}

SCENARIO( "The reduce function obtains max of a list", "[reduce][util]")
{
    GIVEN( "An array of uint32_t values" )
    {
        WHEN( "The array contains 8 unique values" )
        {
            uint32_t input[] = { 1u, 2u, 3u, 4u, 5u, 8u, 7u, 6u };
            auto const n = sizeof( input ) / sizeof( input[ 0 ] );

            THEN( "The reduce function moves the largest to the first index" )
            {
                wrap_reduce{}( input, n );
                REQUIRE( input[ 0 ] == 8u );
            }
        }

        WHEN( "The array is not a power of 2" )
        {
            auto const max = 9u;
            uint32_t input[] = { 1u, 2u, 3u, 4u, 5u, 6u, max };
            auto const n = sizeof( input ) / sizeof( input[ 0 ] );

            THEN( "The reduce function moves the largest to the first index" )
            {
                wrap_reduce{}( input, n );
                REQUIRE( input[ 0 ] == max );
            }
        }

        WHEN( "The array requires 2 warps to cooperate" )
        {
            auto const max = 10u;
            uint32_t input[] = { 1u, 2u, 3u, 4u, 5u, 8u, 7u, 6u,
                                 1u, 2u, 3u, 4u, 5u, 8u, 7u, 6u, 
                                 1u, 2u, 3u, 4u, 5u, 8u, 7u, 6u,
                                 1u, 2u, 3u, 4u, 5u, 8u, 7u, 6u,
                                 1u, 2u, max,4u, 5u, 2u, 7u, 6u };

            auto const n = sizeof( input ) / sizeof( input[ 0 ] );

            THEN( "The reduce function moves the largest to the first index" )
            {
                wrap_reduce{}( input, n );
                REQUIRE( input[ 0 ] == max );
            }
        }
    }
}
