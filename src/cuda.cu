
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>

__device__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__device__ bool match(char * sequence,int i,int j)
{
	if ( ((sequence[i] == 'A' and sequence[j] == 'U' ) or (sequence[i] == 'G' and sequence[j] == 'C' )) and j-i>3)
		return 1;
	return 0;
}

__device__  void printTableGPU(int * T, char* s, int * len1){
	int len = *len1;
	// cout << "\t";
	// s.resize(T.size(),'$');
	printf("\t");
	printf("\n");
	for (int i =0; i<len; i++){
		printf("%c\t", s[i]);
	}
	printf("\n");
	for(int i=0;i<len;i++){
		printf("%c\t", s[i]);
		for(int j=0;j<len;j++){
			printf("%d\t", T[len*i +j]);
		}
		printf("\n");

	}
}



__global__ void mykernel(int * table, char * sequence,int * len1) {
	int len = *len1;

	
	for (int j =1;j<len;j++)
	{
		auto j0 =j;
		for(int i=0;i<len - j0;i++)
		{
			if (i<j)
			{
				int m1 = table[(len*i)+(j-1)];
				int m2 = table[(len*(i+1))+j];
				int m3;
				if(match(sequence,i,j) )
					m3 = table[(len*(i+1))+(j-1)] + match(sequence,i,j);
				else
					m3 = 0;
				int m4 = 0;
				for (int k = i+1;k<j;k++)
				{
					if (table[len*i+k] + table[len*(k+1)+j] > m4)
						m4 = table[len*i+k] + table[len*(k+1)+j];
				}
				//cout << m1 << " " << m2 << " " << m3 << " " << m4 << endl;
				// __vmaxu4 ( unsigned int  a, unsigned int  b )
				

				table[len*i+j] = __vmaxu4(m1,__vmaxu4(m2,__vmaxu4(m3,m4)));
				// if (table[len*i+j] !=0)
				// {
				// 	printf ("%d\t%d\t%d\t%d\t%d\n", m1,m2,m3, m4, table[len*i+j]);
				// }
					
			}
			j++;
		}
		j = j0;
	}
	// printTableGPU(table,sequence,len1);
}

void printTable(int * T, char* s, int * len1){
	int len = *len1;
	// cout << "\t";
	// s.resize(T.size(),'$');
	printf("\t");
	printf("\n");
	for (int i =0; i<len; i++){
		printf("%c\t", s[i]);
	}
	printf("\n");
	for(int i=0;i<len;i++){
		printf("%c\t", s[i]);
		for(int j=0;j<len;j++){
			printf("%d\t", T[len*i +j]);
		}
		printf("\n");

	}
}


int main(void) {
	int size = sizeof(int);
	
	char* seq;
	int a = 15;
	int * len = (int *)malloc(sizeof(int*));
	int * hold_len = len;
	
	len = &a;
	int size2 = (*len) * sizeof(char);

	seq = (char *)malloc( (*len) * sizeof(char)); 
	char * hold_seq = seq;

	for(int i =0;i<(*len);i++){
		if (i<(*len)/2){
			seq[i] = 'G';
		}
		else 
		seq[i] = 'C';
	}
	// seq = "GGAAACACCU";
	// printf("%s", seq);

	
	
	int * x;
	x = (int*)malloc( (*len) * (*len) * sizeof(int) );
	int* hold_x = x;

	int size3 = (*len)*(*len);//(sizeof(x))*sizeof(int);



	for(int i = 0;i<(*len)*(*len);i++){
		x[i] =0;
	}


	int *length = nullptr;
	char *sequence;
	int *table;



	hipMalloc((void **)&length, size);
	hipMalloc((void **)&sequence, size2);
	hipMalloc((void **)&table, size3);

	// Copy inputs to device
	hipMemcpy(length, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(sequence, seq, size2, hipMemcpyHostToDevice);
	hipMemcpy(table, x, size3, hipMemcpyHostToDevice);

	// printTable(x,seq,len);

	//printf("%s", seq);

	mykernel<<<1,1>>>(table, sequence,length);


	hipMemcpy(x, table, size3, hipMemcpyDeviceToHost);

	printTable(x,seq,len);

	



	hipFree(length); hipFree(x); hipFree(sequence);
	free(hold_x);  free(hold_seq); free(hold_len);
	return 0;
}

