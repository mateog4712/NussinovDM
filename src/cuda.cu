#include "hip/hip_runtime.h"
#include "math.h" 	   // sqrtf()
#include <chrono>      // timing library
#include <iostream>
#include <vector>
#include <string>
#include <algorithm>
#include <utility>
#include <type_traits>


// Reuse statically generated data from previous lecture
#include "testCases.hpp"
//#include "../src/simd.hpp"
void nussinovSimd(string sequence){

	//take len of the sequence for further use
	uint16_t len = sequence.length();
	//first we initialize the matrics D
	uint16_t var = 0;
	uint16_t tr1 = 8;
	uint16_t step = 8;
	vector< vector<uint16_t> > table(len+tr1,vector<uint16_t>(len+tr1,var));
	// Load a row of -1's unsigned to be used for xor
	vector<uint16_t> temp = {-1,-1,-1,-1,-1,-1,-1,-1};
	__m128i tempr = _mm_loadu_si128( (__m128i*) &temp[0] );
	
	for (int j = 1;j<len;j++)
	{
		auto j0 = j;
		#pragma omp parallel for 
		for(int i = 0;i<len-j0;i++)
		// for(int i=j-1;i>=0;i--)
		{
			int j = i+j0;
			if (i<j)
			{

				uint16_t m1 = table[i][j-1];
				uint16_t m2 = table[i+1][j];

				auto mB = match(sequence,i,j);
				uint16_t m3 = var;
				
				if(mB)
					m3 = table[i+1][j-1] + mB;
				
				uint16_t m4 = var;

				for (uint16_t k = i+1;k<j;k+=step)
				{	// Load the two rows from the matrix
					__m128i const row1 = _mm_loadu_si128( (__m128i*) &table[i][k] );
					__m128i const row2 = _mm_loadu_si128( (__m128i*) &table[j][k + 1] );
					// add them
					__m128i result_values = _mm_add_epi16(row1,row2);
					
					// xor the results
					__m128i xor_result_values = _mm_xor_si128(result_values, tempr);
					// find the minimum in the results
					__m128i min = _mm_minpos_epu16 (xor_result_values);
					
					uint16_t rez = _mm_extract_epi16(min,0);
					// xor the value back
					rez = rez ^ tempr[0];
					if ( rez > m4){
						m4 = rez;
					}	
						
				}
				// take the maximum of the four operations
				uint16_t ins = max(m1,max(m2,max(m3,m4)));
				table[i][j] = ins;
				table[j][i] = ins;
			}
			// cout <<  << endl;
		}
		// cout << "_____________" << endl;
		j=j0;
	}
	string structure = "";
	uint16_t energy = table[0][len-1];
	// structure = traceback(table, 0, len-1, sequence);
	// cout << energy << endl;
	// cout << structure << endl;
	// printTableFile(table,sequence, "Simd");

bool match(string &sequence,uint16_t i,uint16_t j)
{
	if ( ((sequence[i] == 'A' and sequence[j] == 'U' ) or (sequence[i] == 'G' and sequence[j] == 'C' )) and j-i>3)
		return 1;

	return 0;
}
// This will traceback through the table defined in nussinov
string traceback(vector< vector<uint16_t> > & table, uint16_t i, uint16_t j, string sequence){

	if(i>j)
	{
		return "";
	}

	if(table[i+1][j-1] + 1 == table[i][j] and match(sequence,i,j))
	{

	return "(" + traceback(table,i+1,j-1,sequence) + ")";
	}
	else if(table[i+1][j] == table[i][j])
	{
		return "." + traceback(table,i+1,j,sequence);
	}
	else if(table[i][j-1] == table[i][j])
	{
		return traceback(table,i,j-1,sequence) + ".";
	}
	else{

		for(uint16_t k = i+1; k<j;k++){

			if(table[i][k] + table[k+1][j] == table[i][j]){
			return traceback(table,i,k,sequence) + traceback(table,k+1,j,sequence);		
			}
		}
	}
	return "";
}





}



int main{
/** Definition of Variables and creation of data **/
	
	int numofIter = 5;
	
	for (int j=0;j<numofIter;j++)
	{
		// auto start = chrono::steady_clock::now();
		for(auto i=0;i<testData.size();i++){

			nussinovSimd(testData[i]);

		}
    }
}
